#include "hip/hip_runtime.h"
// resizeWithTexture.cu
#include <hip/hip_runtime.h>
#include <opencv2/core.hpp>
#include <opencv2/core/cuda.hpp>
#include <iostream>

__global__ void resizeKernel(hipTextureObject_t texObj, float* output,
                             int outWidth, int outHeight,
                             int inWidth, int inHeight, int channels)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < outWidth && y < outHeight)
    {
        // Map output pixel (x,y) to input coordinate (u,v)
        float u = (x + 0.5f) * inWidth / outWidth - 0.5f;
        float v = (y + 0.5f) * inHeight / outHeight - 0.5f;
        float4 pixel = tex2D<float4>(texObj, u, v);
        int index = (y * outWidth + x) * channels;
        output[index + 0] = pixel.x;
        output[index + 1] = pixel.y;
        output[index + 2] = pixel.z;
        output[index + 3] = pixel.w;
    }
}

extern "C" void resizeWithTexture(const cv::cuda::GpuMat& input, cv::cuda::GpuMat& output,
                                    int outWidth, int outHeight, hipStream_t stream)
{
    // Get input dimensions.
    int inWidth  = input.cols;
    int inHeight = input.rows;
    int channels = input.channels();  // must be 4 for float4.

    // Create a CUDA array and copy the input into it.
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();
    hipArray_t cuArray;
    hipError_t err = hipMallocArray(&cuArray, &channelDesc, inWidth, inHeight);
    if (err != hipSuccess) {
        std::cerr << "hipMallocArray error: " << hipGetErrorString(err) << std::endl;
        return;
    }

    err = hipMemcpy2DToArray(cuArray, 0, 0, input.ptr(), input.step,
                              inWidth * sizeof(float4), inHeight, hipMemcpyDeviceToDevice);
    if (err != hipSuccess) {
        std::cerr << "hipMemcpy2DToArray error: " << hipGetErrorString(err) << std::endl;
        hipFreeArray(cuArray);
        return;
    }

    // Set up the resource and texture descriptors.
    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cuArray;

    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0]   = hipAddressModeClamp;
    texDesc.addressMode[1]   = hipAddressModeClamp;
    texDesc.filterMode       = hipFilterModeLinear;
    texDesc.readMode         = hipReadModeElementType;
    texDesc.normalizedCoords = 0; // use unnormalized coordinates

    hipTextureObject_t texObj = 0;
    err = hipCreateTextureObject(&texObj, &resDesc, &texDesc, nullptr);
    if (err != hipSuccess) {
        std::cerr << "hipCreateTextureObject error: " << hipGetErrorString(err) << std::endl;
        hipFreeArray(cuArray);
        return;
    }

    // Allocate output GPU memory. Make sure the output GpuMat is CV_32FC4.
    output.create(outHeight, outWidth, input.type());
    float* d_output = reinterpret_cast<float*>(output.ptr());

    dim3 block(16, 16);
    dim3 grid((outWidth + block.x - 1) / block.x, (outHeight + block.y - 1) / block.y);
    resizeKernel<<<grid, block, 0, stream>>>(texObj, d_output, outWidth, outHeight,
                                              inWidth, inHeight, channels);
    hipStreamSynchronize(stream);

    hipDestroyTextureObject(texObj);
    hipFreeArray(cuArray);
}
